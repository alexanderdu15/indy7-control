#include <filesystem>
#include <chrono>
#include <fstream>
#include <thread>
#include <mutex>
#include <atomic>
#include <algorithm>
#include "rclcpp/rclcpp.hpp"
#include "indy7_msgs/msg/joint_state.hpp"
#include "indy7_msgs/msg/joint_trajectory.hpp"
#include "indy7_msgs/msg/joint_trajectory_point.hpp"
#include "trajopt_solver.cuh"
#include "csv_utils.h"

namespace msgs = indy7_msgs::msg;
using Clock = std::chrono::system_clock;
using TimePoint = std::chrono::time_point<Clock>;
using Duration = std::chrono::duration<double>;
using SimTime = rclcpp::Time;

class TrajoptNode : public rclcpp::Node
{
public:
    TrajoptNode(const std::string& traj_file)
    : Node("trajopt_node"), 
      timestep_(std::chrono::duration<double>(0.01)),
      pcg_exit_tol_(5e-4),
      pcg_max_iter_(173),
      state_updated_(false),
      warm_start_complete_(false),
      trajectory_start_time_(),
      use_sim_time_(false)
    {
        use_sim_time_ = get_parameter("use_sim_time").as_bool();
        RCLCPP_INFO(this->get_logger(), "Using %s time", use_sim_time_ ? "simulation" : "system");

        RCLCPP_INFO(this->get_logger(), "Initializing TrajoptNode");
        state_sub_ = create_subscription<msgs::JointState>(
            "joint_states", 1, 
            std::bind(&TrajoptNode::stateCallback, this, std::placeholders::_1)
        );

        traj_pub_ = create_publisher<msgs::JointTrajectory>("joint_trajectory", 1);
        
        // Initialize solver with trajectory from file
        std::vector<float> goal_eePos_traj_1d = readCsvToVector<float>(traj_file);
        solver_ = std::make_unique<TrajoptSolver<float>>(
            goal_eePos_traj_1d,
            timestep_.count(),
            pcg_exit_tol_,
            pcg_max_iter_
        );
        RCLCPP_INFO(this->get_logger(), "Solver initialized");

        // Pre-allocate
        traj_msg_.knot_points = solver_->numKnotPoints();
        for (int i = 0; i < solver_->numKnotPoints(); i++) {
            msgs::JointTrajectoryPoint point;
            traj_msg_.points.emplace_back(point);
        }
        full_state_.reserve(solver_->stateSize());

        
        RCLCPP_INFO(this->get_logger(), "Waiting for initial state...");
        while (rclcpp::ok() && !state_updated_) {
            rclcpp::spin_some(this->get_node_base_interface());
            std::this_thread::sleep_for(std::chrono::milliseconds(10));
        }

        std::vector<float> current_joint_positions(current_state_.positions.begin(), current_state_.positions.end());
        solver_->initializeXU(current_joint_positions);
        RCLCPP_INFO(this->get_logger(), "Received initial state, starting solver warm start...");
        solver_->warmStart();
        hipDeviceSynchronize(); //cuda operations are asynchronous, so we need to synchronize here
        warm_start_complete_ = true;
        if (use_sim_time_) {
            // Wait for valid sim time
            while (rclcpp::ok() && current_state_.header.stamp.sec == 0) {
                rclcpp::spin_some(this->get_node_base_interface());
                std::this_thread::sleep_for(std::chrono::milliseconds(10));
            }
            trajectory_start_stamp_ = current_state_.header.stamp;
        } else {
            trajectory_start_time_ = Clock::now();
        }
        RCLCPP_INFO(this->get_logger(), "Warm start complete");
    }

    ~TrajoptNode() {
        RCLCPP_INFO(this->get_logger(), "Shutting down TrajoptNode");
    }

private:
    void stateCallback(const std::shared_ptr<const msgs::JointState>& msg)
    {
        // Skip if optimization is already running or warm start is not complete
        if (optimization_in_progress_.load()) {
            return;
        }

        {
            std::lock_guard<std::mutex> lock(state_mutex_);
            current_state_ = *msg;
            // Convert from degrees to radians
            for (int i = 0; i < 6; i++) {
                current_state_.positions[i] = current_state_.positions[i];
                current_state_.velocities[i] = current_state_.velocities[i];
            }
            state_updated_ = true;
        }

        if (!warm_start_complete_) {
            return;
        }

        if (solver_->isTrajectoryComplete()) {
            return;
        }

        optimization_in_progress_ = true;

        full_state_.clear();  // Clear before inserting new state
        full_state_.insert(full_state_.end(), 
            current_state_.positions.begin(), 
            current_state_.positions.end());
        full_state_.insert(full_state_.end(), 
            current_state_.velocities.begin(), 
            current_state_.velocities.end());

        // Calculate elapsed time based on use_sim_time_ setting
        double elapsed_time;
        if (use_sim_time_) {
            SimTime current_time(msg->header.stamp);
            SimTime start_time(trajectory_start_stamp_);
            elapsed_time = (current_time - start_time).seconds();
        } else {
            auto now = Clock::now();
            Duration elapsed = now - trajectory_start_time_;
            elapsed_time = elapsed.count();
        }
        
        // Shift trajectory and run optimization
        solver_->shiftTrajectory(full_state_, elapsed_time);
        std::string stats = solver_->runTrajoptIteration();
        RCLCPP_INFO(this->get_logger(), "Optimization stats: %s", stats.c_str());
        
        RCLCPP_INFO(this->get_logger(), "Trajectory offset: %u", solver_->getTrajectoryOffset());

        publishTrajectory(msg->header.stamp);

        optimization_in_progress_ = false;
    }

    void publishTrajectory(const builtin_interfaces::msg::Time& stamp)
    {
        auto traj_msg = msgs::JointTrajectory();
        traj_msg.header.stamp = stamp;
        traj_msg.knot_points = solver_->numKnotPoints(); 
        traj_msg.dt = timestep_.count();  // Convert Duration to double

        // Get optimized trajectory from solver
        const auto [traj_data, traj_size] = solver_->getOptimizedTrajectory();
        
        const int stride = solver_->stateSize() + solver_->controlSize();
        
        for (int i = 0; i < traj_msg.knot_points; i++) {
            for (size_t j = 0; j < 6; ++j) {
                traj_msg.points[i].positions[j] = traj_data[i * stride + j];// * 180.0 / M_PI;
                traj_msg.points[i].velocities[j] = traj_data[i * stride + solver_->stateSize()/2 + j];// * 180.0 / M_PI;
                traj_msg.points[i].torques[j] = traj_data[i * stride + solver_->stateSize() + j];
            }
        }

        traj_pub_->publish(traj_msg);
    }

    std::unique_ptr<TrajoptSolver<float, 12, 6, 128, 128>> solver_;
    rclcpp::Subscription<msgs::JointState>::SharedPtr state_sub_;
    rclcpp::Publisher<msgs::JointTrajectory>::SharedPtr traj_pub_;

    const Duration timestep_;
    const float pcg_exit_tol_;
    const int pcg_max_iter_;

    msgs::JointState current_state_;
    std::atomic<bool> state_updated_;
    std::mutex state_mutex_;
    TimePoint trajectory_start_time_;
    std::atomic<bool> warm_start_complete_;
    std::atomic<bool> optimization_in_progress_{false};

    msgs::JointTrajectory traj_msg_;
    std::vector<float> full_state_;

    bool use_sim_time_;
    builtin_interfaces::msg::Time trajectory_start_stamp_;
};

int main(int argc, char* argv[])
{
    rclcpp::init(argc, argv);
    auto node = std::make_shared<TrajoptNode>(argv[1]);
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}